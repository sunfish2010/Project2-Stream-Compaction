#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer() {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int N, int *odata, int d){
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index < N && (index % (1 << (d + 1)) == 0) ){
                odata[index + (1 << (d + 1)) - 1] += odata[index + (1 << d) - 1];
            }
        }


        __global__ void kernDownSweep(int N, int *odata, int d){
             int index = threadIdx.x + (blockIdx.x * blockDim.x);
             if (index < N && (index % (1 << (d + 1)) == 0) ) {
                int tmp = odata[index + (1 << d) - 1];
                odata[index + (1 << d) - 1]; = odata[index + (1 << (d + 1)) - 1];
                odata[index + (1 << (d + 1)) - 1] += tmp;
             }

        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // when n is not power of two, need to allocate more space to zero pad
            int d = ilog2ceil(n);
            int N = 1 << d;

            dim3 fullBlockPerGrid;
            int* dev_out;

            hipMalloc((void**)&dev_out, sizeof(int) * N);
            checkCUDAError("hipMalloc dev_out failed");

            hipMemset(dev_out, 0, sizeof(int));
            checkCUDAError("cuda Memset failed");

            hipMemcpy(dev_out, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpyHostToDevice failed");


            timer().startGpuTimer();
            // without shared memory, the algorithm needs to be called for d times
            for (int i = 0; i < d; i++){
                fullBlockPerGrid = (1 << (d - i - 1) + blockSize - 1) / blockSize;
                kernUpSweep<<<fullBlockPerGrid, blockSize>>>(N, dev_out, i);
                checkCUDAError("kernUpSweep failed");
            }

            hipMemset(dev_out + N - 1, 0, sizeof(int));
            for (int i = d - 1; i >= 0; i--){
                fullBlockPerGrid = (1 << (d - i - 1) + blockSize - 1) / blockSize;
                kernDownSweep<<<fullBlockPerGrid, blockSize>>>(N, dev_out, i);
                checkCUDAError("kernDownpSweep failed");
            }

            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpyDeviceToHost failed");

            hipFree(odata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {

            dim3 fullBlockPerGrid((n + blockSize - 1) / blockSize);
            int* bools, *indices, *dev_in, *dev_out;
            int num_element;

            hipMalloc((void**)&bools, sizeof(int) * n);
            checkCUDAError("hipMalloc bools failed");
            hipMalloc((void**)&indices, sizeof(int) * n);
            checkCUDAError("hipMalloc indices failed");
            hipMalloc((void**)&dev_out, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_out failed");
            hipMalloc((void**)&dev_in, sizeof(int) * n);
            checkCUDAError("hipMalloc dev_in failed");

            // lots of memcpy...

            hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpyHostToDevice failed");

            timer().startGpuTimer();
            StreamCompaction::Common:: kernMapToBoolean<<<fullBlockPerGrid, blockSize>>>(n, bools, dev_in);
            checkCUDAError("kernMapToBoolean failed");

            hipMemcpy(odata, bools, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpyDeviceToHost failed");

            scan(n, odata, odata);
            num_element = odata[n - 1];

            hipMemcpy(indices, odata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpyHostToDevice failed");

            StreamCompaction::Common::kernScatter(n, dev_out, dev_in, bools, indices);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpyDeviceToHost failed");

            hipFree(bools);
            hipFree(indices);
            hipFree(dev_in);
            hipFree(dev_out);

            return num_element;
        }
    }
}
